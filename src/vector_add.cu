#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#define assert(x) \
    if (!x) exit(1)
__global__ void vector_add(const float *a, const float *b, float * c, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        c[i] = a[i] + b[i];
    }
}

int main(void) {
    hipError_t err = hipSuccess;
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);
    if (h_a == nullptr || h_b == nullptr || h_c == nullptr)
        return 1;
    for (int i = 0; i < numElements; ++i) {
        h_a[i] = i;
        h_b[i] = i;
    }
    float *d_a = nullptr;
    err = hipMalloc((void**)&d_a, size);
    assert(err == hipSuccess); 
    float *d_b = nullptr;
    err = hipMalloc((void**)&d_b, size);
    assert(err == hipSuccess);
    float *d_c = nullptr;
    err = hipMalloc((void**)&d_c, size);
    assert(err == hipSuccess);
    
    err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    assert(err == hipSuccess);
    err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    assert(err == hipSuccess);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, numElements);
    err = hipGetLastError();
    assert(err == hipSuccess);
    
    err = hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    assert(err == hipSuccess);
    
    for (int i = 0; i < numElements; ++i) {
        assert(fabs(h_a[i] + h_b[i] - h_c[i]) < 1e-5);
    }
    err = hipFree(d_a);
    err = hipFree(d_b);
    err = hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}
